#include "hip/hip_runtime.h"
#define MAX_SIZE 128
#define NUM_DIMENSIONS 3
#define MAX_RESULT_SIZE 512
#define RANGE 0.2f;
#define BLOCKSIZE 256
#include "hip/hip_runtime.h"
#include <stdlib.h>

typedef struct
{
    
    unsigned int data[MAX_SIZE];
    unsigned int front;
    unsigned int tail;
    unsigned int size;
} Queue;

__device__ bool push_back(Queue* queue, unsigned int index)
{
    if (queue->size < MAX_SIZE)
    {
        queue->data[queue->tail] = index;
        queue->tail = (queue->tail + 1) % MAX_SIZE;
        queue->size++;
        return true;
    }
    return false;
    
}

__device__ unsigned int pop_front(Queue* queue)
{
    if (queue->size > 0)
    {
        unsigned int element = queue->data[queue->front];
        queue->front = (queue->front + 1) % MAX_SIZE;
        queue->size--;
        return element;
    }
}

__device__ void erase_first_n_elements(Queue* queue, unsigned int n)
{
    unsigned int elementsToErase = queue->size - n > 0 ? n : queue->size;
    queue->size -=elementsToErase;
    queue->front = (queue->front + elementsToErase) % MAX_SIZE;
    
}


__device__ unsigned int leftSonIndex(unsigned int index)
{
    return 2 * index + 1;
}


__device__ unsigned int rightSonIndex(unsigned int index)
{
    return 2 * index + 2;
}


__device__ bool intersects(unsigned int index,  float* theDimensions, unsigned int nPoints,
                           float* minPoint, float* maxPoint, int dimension)
{
    return (theDimensions[nPoints * dimension + index] <= maxPoint[dimension]
            && theDimensions[nPoints * dimension + index] >= minPoint[dimension]);
}


__device__ bool isInTheBox(unsigned int index,  float* theDimensions, unsigned int nPoints,
                           float* minPoint, float* maxPoint)
{
    bool inTheBox = true;
    for (int i = 0; i < NUM_DIMENSIONS; ++i)
    {
        inTheBox &= (theDimensions[nPoints * i + index] <= maxPoint[i]
                     && theDimensions[nPoints * i + index] >= minPoint[i]);
    }
    
    return inTheBox;
}


__global__ void CUDASearchInTheKDBox(unsigned int nPoints,  float* dimensions,  unsigned int* ids,  unsigned int* results)
{
    
    // Global Thread ID
    unsigned int point_index = blockIdx.x*blockDim.x+threadIdx.x;
    
    //	float range = 0.1f;
    if(point_index < nPoints)
    {
        
        int theDepth = floor(log2((float)nPoints));
        float minPoint[NUM_DIMENSIONS];
        float maxPoint[NUM_DIMENSIONS];
        for(int i = 0; i<NUM_DIMENSIONS; ++i)
        {
            minPoint[i] = dimensions[nPoints*i+point_index] - RANGE;
            maxPoint[i] = dimensions[nPoints*i+point_index] + RANGE;
        }
        
        Queue indecesToVisit;
        indecesToVisit.front = indecesToVisit.tail =indecesToVisit.size =0;
        unsigned int pointsFound=0;
        unsigned int resultIndex = nPoints + MAX_RESULT_SIZE*point_index;
        push_back(&indecesToVisit, 0);
        
        for (int depth = 0; depth < theDepth + 1; ++depth)
        {
            int dimension = depth % NUM_DIMENSIONS;
            unsigned int numberOfIndecesToVisitThisDepth =
            indecesToVisit.size;
            
            for (unsigned int visitedIndecesThisDepth = 0;
                 visitedIndecesThisDepth < numberOfIndecesToVisitThisDepth;
                 visitedIndecesThisDepth++)
            {
                
                //				unsigned int index = indecesToVisit.data[(indecesToVisit.front+visitedIndecesThisDepth)% MAX_SIZE];
                unsigned int index = pop_front(&indecesToVisit);
                //				if(point_index == 0)
                //				{
                //					printf("index: %d, dimensions: %f %f %f\n", index, dimensions[index], dimensions[nPoints+index], dimensions[2*nPoints+index]);
                //				}
                
                bool intersection = intersects(index,dimensions, nPoints, minPoint, maxPoint,
                                               dimension);
                
                if(intersection && isInTheBox(index, dimensions, nPoints, minPoint, maxPoint))
                {
                    if(pointsFound < MAX_RESULT_SIZE)
                    {
                        //						if(point_index == 0)
                        //						{
                        //							printf("index: %d added to results", index);
                        //						}
                        
                        results[resultIndex] = index;
                        resultIndex++;
                        pointsFound++;
                    }
                    
                }
                
                bool isLowerThanBoxMin = dimensions[nPoints*dimension + index]
                < minPoint[dimension];
                int startSon = isLowerThanBoxMin; //left son = 0, right son =1
                
                int endSon = isLowerThanBoxMin || intersection;
                
                for (int whichSon = startSon; whichSon < endSon + 1; ++whichSon)
                {
                    unsigned int indexToAdd = leftSonIndex(index) + whichSon;
                    
                    if (indexToAdd < nPoints)
                    {
                        push_back(&indecesToVisit,indexToAdd);
                    
                    }
                }
            }
            
            //			erase_first_n_elements(&indecesToVisit,numberOfIndecesToVisitThisDepth );
        }
        
        results[point_index] = pointsFound;
        
    }
    
}

#define MAX_FRONTIER 256

__global__ void CUDABFSparallel (unsigned int nPoints,  float* dimensions,  unsigned int* ids,  unsigned int* results) {
	__shared__ int frontier[MAX_FRONTIER];
	__shared__ int frontnext[MAX_FRONTIER];
	__shared__ int pfound[MAX_FRONTIER];

	unsigned int point_index = blockIdx.x;
	unsigned int thread_index = threadIdx.x;

	unsigned int found = 0;

	if (point_index < nPoints) {
		int theDepth = floor(log2((float)nPoints));
		float minPoint[NUM_DIMENSIONS];
		float maxPoint[NUM_DIMENSIONS];
        for(int i = 0; i<NUM_DIMENSIONS; ++i) {
			minPoint[i] = dimensions[nPoints*i+point_index] - RANGE;
			maxPoint[i] = dimensions[nPoints*i+point_index] + RANGE;
		}

		//if (thread_index < MAX_FRONTIER) {
			frontier[0] = 0;

			int startSon;
			int endSon = startSon-1;

			for (int depth = 0; depth < theDepth+1; ++depth) {
				int dimension = depth % NUM_DIMENSIONS;

				unsigned int index = frontier[thread_index];
				frontier[thread_index] = nPoints;

				if (index < nPoints) {
					int intersection = intersects(index, dimensions,nPoints,minPoint,maxPoint,dimension);

					if (intersection && isInTheBox(index,dimensions,nPoints,minPoint,maxPoint)) {
						if (found < MAX_RESULT_SIZE) {
								//results[resultIndex] = index;
								resultIndex++;
								found++;
						}
					}

					startSon = dimensions[nPoints*dimension+index] < minPoint[dimension];
					endSon = startSon || intersection;
				}

				int next = 0;

				for (int whichSon = startSon; whichSon < endSon+1; ++whichSon)
					next += (leftSonIndex(index) + whichSon < nPoints);
				frontnext[thread_index] = next;

				__syncthreads();

				thrust::exclusive_scan(frontnext,frontnext+MAX_FRONTIER,frontnext);				
				write_offset = frontnext[thread_index];

				for (int i = startSon; i < startSon+next; i++, write_offset++) {
					frontier[write_offset] = leftSonIndex(index)+i;
				}

				__syncthreads();

				endSon = startSon-1;
			}

			pfound[thread_index] = found;

			unsigned int pointsFound = thrust::reduce(pfound, pfound+MAX_FRONTIER);

			results[point_index] = pointsFound;
		//}
	}
}

void CUDAKernelWrapper(unsigned int nPoints,float *d_dim,unsigned int *d_ids,unsigned int *d_results)
{


    // Number of thread blocks
    unsigned int gridSize = (int)ceil((float)nPoints/BLOCKSIZE);
    
    CUDASearchInTheKDBox<<<gridSize, BLOCKSIZE>>>(nPoints, d_dim, d_ids,d_results);


    
}

